#include "hip/hip_runtime.h"
#include "gpu_memcpy.h"

__global__ void gpu_print(short *a, int size) {
  for(int i = 0; i < size/2; ++i) printf("d_a[%d]: %d\n", i, a[i]);
}
void CUDART_CB my_callback(hipStream_t stream, hipError_t status, void* data)
{
    printf("callback from stream %d\n", *((int*)data));
}
// int memcpy_host2gpu(short* gpu_dst, short* host_src, size_t size) {
short* memcpy_host2gpu(short* host_src, int size) {    

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK_CUDA(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using device %d: %s\n", dev, deviceProp.name);
    CHECK_CUDA(hipSetDevice(dev));


  short *d_a;
  CHECK_CUDA(hipMalloc(&d_a, size));
  int stream_ids[1];
  stream_ids[0] = 0;
  hipStream_t *streams = (hipStream_t*)malloc(sizeof(hipStream_t));
  CHECK_CUDA(hipStreamCreate(&streams[0]));
  CHECK_CUDA(hipStreamAddCallback(streams[0], my_callback, (void*)stream_ids, 0));

  CHECK_CUDA(hipDeviceSynchronize());
  // CHECK_CUDA(hipHostRegister(host_src, size, hipHostRegisterDefault));
  // CHECK_CUDA(hipHostRegister(host_src, size, hipHostRegisterMapped));
  CHECK_CUDA(hipHostRegister(host_src, size, hipHostRegisterPortable));
  // CHECK_CUDA(hipHostRegister((void*)host_src, size, hipHostRegisterIoMemory));
  getchar();
  
  short* host_src_d;
  CHECK_CUDA(hipHostGetDevicePointer((void**)&host_src_d, host_src, 0));
  
  // for(int i = 0; i < 100; ++i) printf("test_register[%d]: %d\n", i, test_register[i]);

  {
    CHECK_CUDA(hipMemcpy(d_a, host_src_d, size, hipMemcpyHostToDevice));
    // CHECK_CUDA(hipMemcpy(host_src, d_a, size, hipMemcpyDeviceToHost));
    gpu_print<<<1, 1>>>(d_a, size);
    CHECK_CUDA(hipDeviceSynchronize());
  }
  printf("gpu addr: %x\n", d_a);
  // getchar();
  // free host registered space?
  CHECK_CUDA(hipHostUnregister(host_src));
  return d_a;
}

short* memcpy_gpu2host(short* Host_dst, short* GPU_dst, int size) {
  CHECK_CUDA(hipHostRegister(Host_dst, size, hipHostRegisterIoMemory));

  short* host_dst_d;
  CHECK_CUDA(hipHostGetDevicePointer((void**)&host_dst_d, Host_dst, 0));


  {
    CHECK_CUDA(hipMemcpy(host_dst_d, GPU_dst, size, hipMemcpyDeviceToHost));
    // gpu_print<<<1, 1>>>(d_a, size);
    CHECK_CUDA(hipDeviceSynchronize());
  }
  // getchar();
  return NULL;
}